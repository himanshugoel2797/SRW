
#include <hip/hip_runtime.h>
/************************************************************************//**
 * File: sroptdrf_gpu.cu
 * Description: Optical element: Zone Plate (CUDA implementation)
 * Project: Synchrotron Radiation Workshop
 * First release: 2024
 *
 * Copyright (C) Brookhaven National Laboratory
 * All Rights Reserved
 *
 * @author H.Goel
 * @version 1.0
 ***************************************************************************/

#ifdef _OFFLOAD_GPU
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "math_constants.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
#include "sroptzp.h"

//Implementation of the RadPointModifier's GPU function for the srTRectAperture class
int srTZonePlate::RadPointModifierParallel(srTSRWRadStructAccessData* pRadAccessData, void* pBufVars, long pBufVarsSz, TGPUUsageArg *pGPU) 
{ 
    return RadPointModifierParallelImpl<srTZonePlate>(pRadAccessData, pBufVars, pBufVarsSz, this, pGPU); 
}

#endif