
#include <hip/hip_runtime.h>
/************************************************************************//**
 * File: sroptelm_gpu.cu
 * Description: Optical element (general CUDA functions)
 * Project: Synchrotron Radiation Workshop
 * First release: 2023
 *
 * Copyright (C) Brookhaven National Laboratory
 * All Rights Reserved
 *
 * @author H.Goel
 * @version 1.0
 ***************************************************************************/

#ifdef _OFFLOAD_GPU
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "math_constants.h"
#include "cooperative_groups.h" //HG31072024
#include "cooperative_groups/reduce.h" //HG31072024

#include <stdio.h>
#include <iostream>
#include <chrono>
#include "sroptelm.h"
#include "sroptelm_gpu.h"

namespace cg = cooperative_groups; //HG31072024

//__global__ void TreatStronglyOscillatingTerm_Kernel(srTSRWRadStructAccessData RadAccessData, bool TreatPolCompX, bool TreatPolCompZ, double ConstRx, double ConstRz, int ieStart) 
__global__ void TreatStronglyOscillatingTerm_Kernel(srTSRWRadStructAccessData* pRadAccessData, bool TreatPolCompX, bool TreatPolCompZ, double ConstRx, double ConstRz, int ieStart, int ieBefEnd) //HG27072024
{
    int ie = (blockIdx.x * blockDim.x + threadIdx.x); //ne range
	int ix = (blockIdx.y * blockDim.y + threadIdx.y); //nx range
	int iz = (blockIdx.z * blockDim.z + threadIdx.z); //nz range
	
    if (ix < pRadAccessData->nx && iz < pRadAccessData->nz && ie < ieBefEnd) 
    {
        double ePh = pRadAccessData->eStart + pRadAccessData->eStep * (ie - ieStart);
        if (pRadAccessData->PresT == 1)
        {
            ePh = pRadAccessData->avgPhotEn; //?? OC041108
        }

        double ConstRxE = ConstRx * ePh;
        double ConstRzE = ConstRz * ePh;
        if (pRadAccessData->Pres == 1)
        {
            //double Lambda_m = 1.239854e-06/ePh;
            double Lambda_m = 1.239842e-06 / ePh;
            if (pRadAccessData->PhotEnergyUnit == 1) Lambda_m *= 0.001; // if keV

            double Lambda_me2 = Lambda_m * Lambda_m;
            ConstRxE *= Lambda_me2;
            ConstRzE *= Lambda_me2;
        }

        double z = (pRadAccessData->zStart - pRadAccessData->zc) + (iz * pRadAccessData->zStep);
        double PhaseAddZ = 0;
        if (pRadAccessData->WfrQuadTermCanBeTreatedAtResizeZ) PhaseAddZ = ConstRzE * z * z;

        double x = (pRadAccessData->xStart - pRadAccessData->xc) + (ix * pRadAccessData->xStep);
        double Phase = PhaseAddZ;
        if (pRadAccessData->WfrQuadTermCanBeTreatedAtResizeX) Phase += ConstRxE * x * x;

        float SinPh, CosPh;
        sincosf(Phase, &SinPh, &CosPh);

        long long PerX = pRadAccessData->ne << 1;
        long long PerZ = PerX * pRadAccessData->nx;
        long long offset = ie * 2 + iz * PerZ + ix * PerX;
        
		if (TreatPolCompX)
		{
			float* pExRe = pRadAccessData->pBaseRadX + offset;
			float* pExIm = pExRe + 1;
			double ExReNew = (*pExRe) * CosPh - (*pExIm) * SinPh;
			double ExImNew = (*pExRe) * SinPh + (*pExIm) * CosPh;
			*pExRe = (float)ExReNew; *pExIm = (float)ExImNew;
		}
		if (TreatPolCompZ)
		{
			float* pEzRe = pRadAccessData->pBaseRadZ + offset;
			float* pEzIm = pEzRe + 1;
			double EzReNew = (*pEzRe) * CosPh - (*pEzIm) * SinPh;
			double EzImNew = (*pEzRe) * SinPh + (*pEzIm) * CosPh;
			*pEzRe = (float)EzReNew; *pEzIm = (float)EzImNew;
		}
    }
}

void srTGenOptElem::TreatStronglyOscillatingTerm_GPU(srTSRWRadStructAccessData& RadAccessData, bool TreatPolCompX, bool TreatPolCompZ, double ConstRx, double ConstRz, int ieStart, int ieBefEnd, TGPUUsageArg* pGPU)
{
	if (RadAccessData.pBaseRadX != NULL)
	{
		RadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, RadAccessData.pBaseRadX, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
		CAuxGPU::EnsureDeviceMemoryReady(pGPU, RadAccessData.pBaseRadX);
	}
	if (RadAccessData.pBaseRadZ != NULL)
	{
		RadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, RadAccessData.pBaseRadZ, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
		CAuxGPU::EnsureDeviceMemoryReady(pGPU, RadAccessData.pBaseRadZ);
	}

	srTSRWRadStructAccessData* pRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &RadAccessData, sizeof(srTSRWRadStructAccessData)); //HG27072024
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData_dev);

    const int bs = 256;
	dim3 blocks(ieBefEnd - ieStart, RadAccessData.nx, RadAccessData.nz);
	dim3 threads(1, 1, 1);
	if (blocks.x > 1)
	{
		if (blocks.x >= bs)
			threads.x = bs;
		else
			threads.x = blocks.x;


		if (blocks.x % threads.x > 0) blocks.x = blocks.x / threads.x + 1;
		else blocks.x = blocks.x / threads.x;

		if (bs / threads.x > 1)
		{
			threads.y = bs / threads.x;
			if (blocks.y % threads.y > 0) blocks.y = blocks.y / threads.y + 1;
			else blocks.y = blocks.y / threads.y;
		}
	}
	else
	{
		if (bs / blocks.y > 1)
		{
			threads.y = blocks.y;
			blocks.y = 1;
			threads.z = bs / threads.y;
			if (blocks.z % threads.z > 0) blocks.z = blocks.z / threads.z + 1;
			else blocks.z = blocks.z / threads.z;
		}
		else
		{
			threads.y = bs;
			if (blocks.y % threads.y > 0) blocks.y = blocks.y / threads.y + 1;
			else blocks.y = blocks.y / threads.y;
		}
	}

    //TreatStronglyOscillatingTerm_Kernel<< <blocks, threads >> > (RadAccessData, TreatPolCompX, TreatPolCompZ, ConstRx, ConstRz, ieStart);
    TreatStronglyOscillatingTerm_Kernel<< <blocks, threads >> > (pRadAccessData_dev, TreatPolCompX, TreatPolCompZ, ConstRx, ConstRz, ieStart, ieBefEnd); //HG27072024

	CAuxGPU::ToHostAndFree(pGPU, pRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true); //HG27072024
	
	CAuxGPU::MarkUpdated(pGPU, RadAccessData.pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, RadAccessData.pBaseRadZ, true, false);

//#ifndef _DEBUG
	if (RadAccessData.pBaseRadX != NULL)
		RadAccessData.pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, RadAccessData.pBaseRadX);
	if (RadAccessData.pBaseRadZ != NULL)
		RadAccessData.pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, RadAccessData.pBaseRadZ);
//#endif

//#ifdef _DEBUG
//	if (RadAccessData.pBaseRadX != NULL)
//		RadAccessData.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, RadAccessData.pBaseRadX, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
//	if (RadAccessData.pBaseRadZ != NULL)
//		RadAccessData.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, RadAccessData.pBaseRadZ, 2*RadAccessData.ne*RadAccessData.nx*RadAccessData.nz*sizeof(float));
//	cudaStreamSynchronize(0);
//	auto err = cudaGetLastError();
//	printf("%s\r\n", cudaGetErrorString(err));
//#endif
}

__global__ void MakeWfrEdgeCorrection_Kernel(srTSRWRadStructAccessData* pRadAccessData, float* pDataEx, float* pDataEz, srTDataPtrsForWfrEdgeCorr DataPtrs, float dxSt, float dxFi, float dzSt, float dzFi)
{
    int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
    int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range

    if (ix < pRadAccessData->nx && iz < pRadAccessData->nz)
    {
		//float dxSt = (float)DataPtrs.dxSt;
		//float dxFi = (float)DataPtrs.dxFi;
		//float dzSt = (float)DataPtrs.dzSt;
		//float dzFi = (float)DataPtrs.dzFi;
		float dxSt_dzSt = dxSt * dzSt;
		float dxSt_dzFi = dxSt * dzFi;
		float dxFi_dzSt = dxFi * dzSt;
		float dxFi_dzFi = dxFi * dzFi;

		//long TwoNz = pRadAccessData->nz << 1; //OC25012024 (commented-out)
		long PerX = 2;
		long PerZ = PerX * pRadAccessData->nx;

        float fSSExRe = DataPtrs.fxStzSt[0];
        float fSSExIm = DataPtrs.fxStzSt[1];
        float fSSEzRe = DataPtrs.fxStzSt[2];
        float fSSEzIm = DataPtrs.fxStzSt[3];
        
        float fFSExRe = DataPtrs.fxFizSt[0];
        float fFSExIm = DataPtrs.fxFizSt[1];
        float fFSEzRe = DataPtrs.fxFizSt[2];
        float fFSEzIm = DataPtrs.fxFizSt[3];
        
        float fSFExRe = DataPtrs.fxStzFi[0];
        float fSFExIm = DataPtrs.fxStzFi[1];
        float fSFEzRe = DataPtrs.fxStzFi[2];
        float fSFEzIm = DataPtrs.fxStzFi[3];
        
        float fFFExRe = DataPtrs.fxFizFi[0];
        float fFFExIm = DataPtrs.fxFizFi[1];
        float fFFEzRe = DataPtrs.fxFizFi[2];
        float fFFEzIm = DataPtrs.fxFizFi[3];

		float bRe, bIm, cRe, cIm;

		long long Two_iz = iz << 1;
		long long Two_iz_p_1 = Two_iz + 1;
		long long Two_ix = ix << 1;
		long long Two_ix_p_1 = Two_ix + 1;

		float* tEx = pDataEx + iz * PerZ + ix * PerX, * tEz = pDataEz + iz * PerZ + ix * PerX;
		float ExRe = *tEx, ExIm = *(tEx + 1);
		float EzRe = *tEz, EzIm = *(tEz + 1);

		if (dxSt != 0.f)
		{
			float ExpXStRe = DataPtrs.ExpArrXSt[Two_ix], ExpXStIm = DataPtrs.ExpArrXSt[Two_ix_p_1];

			bRe = DataPtrs.FFTArrXStEx[Two_iz]; bIm = DataPtrs.FFTArrXStEx[Two_iz_p_1];
			ExRe += (float)(dxSt * (ExpXStRe * bRe - ExpXStIm * bIm));
			ExIm += (float)(dxSt * (ExpXStRe * bIm + ExpXStIm * bRe));

			bRe = DataPtrs.FFTArrXStEz[Two_iz]; bIm = DataPtrs.FFTArrXStEz[Two_iz_p_1];
			EzRe += (float)(dxSt * (ExpXStRe * bRe - ExpXStIm * bIm));
			EzIm += (float)(dxSt * (ExpXStRe * bIm + ExpXStIm * bRe));

			if (dzSt != 0.f)
			{
				bRe = DataPtrs.ExpArrZSt[Two_iz], bIm = DataPtrs.ExpArrZSt[Two_iz_p_1];
				cRe = ExpXStRe * bRe - ExpXStIm * bIm; cIm = ExpXStRe * bIm + ExpXStIm * bRe;

				ExRe += (float)(dxSt_dzSt * (fSSExRe * cRe - fSSExIm * cIm));
				ExIm += (float)(dxSt_dzSt * (fSSExRe * cIm + fSSExIm * cRe));
				EzRe += (float)(dxSt_dzSt * (fSSEzRe * cRe - fSSEzIm * cIm));
				EzIm += (float)(dxSt_dzSt * (fSSEzRe * cIm + fSSEzIm * cRe));
			}
			if (dzFi != 0.f)
			{
				bRe = DataPtrs.ExpArrZFi[Two_iz], bIm = DataPtrs.ExpArrZFi[Two_iz_p_1];
				cRe = ExpXStRe * bRe - ExpXStIm * bIm; cIm = ExpXStRe * bIm + ExpXStIm * bRe;

				ExRe -= (float)(dxSt_dzFi * (fSFExRe * cRe - fSFExIm * cIm));
				ExIm -= (float)(dxSt_dzFi * (fSFExRe * cIm + fSFExIm * cRe));
				EzRe -= (float)(dxSt_dzFi * (fSFEzRe * cRe - fSFEzIm * cIm));
				EzIm -= (float)(dxSt_dzFi * (fSFEzRe * cIm + fSFEzIm * cRe));
			}
		}
		if (dxFi != 0.f)
		{
			float ExpXFiRe = DataPtrs.ExpArrXFi[Two_ix], ExpXFiIm = DataPtrs.ExpArrXFi[Two_ix_p_1];

			bRe = DataPtrs.FFTArrXFiEx[Two_iz]; bIm = DataPtrs.FFTArrXFiEx[Two_iz_p_1];
			ExRe -= (float)(dxFi * (ExpXFiRe * bRe - ExpXFiIm * bIm));
			ExIm -= (float)(dxFi * (ExpXFiRe * bIm + ExpXFiIm * bRe));

			bRe = DataPtrs.FFTArrXFiEz[Two_iz]; bIm = DataPtrs.FFTArrXFiEz[Two_iz_p_1];
			EzRe -= (float)(dxFi * (ExpXFiRe * bRe - ExpXFiIm * bIm));
			EzIm -= (float)(dxFi * (ExpXFiRe * bIm + ExpXFiIm * bRe));

			if (dzSt != 0.f)
			{
				bRe = DataPtrs.ExpArrZSt[Two_iz], bIm = DataPtrs.ExpArrZSt[Two_iz_p_1];
				cRe = ExpXFiRe * bRe - ExpXFiIm * bIm; cIm = ExpXFiRe * bIm + ExpXFiIm * bRe;

				ExRe -= (float)(dxFi_dzSt * (fFSExRe * cRe - fFSExIm * cIm));
				ExIm -= (float)(dxFi_dzSt * (fFSExRe * cIm + fFSExIm * cRe));
				EzRe -= (float)(dxFi_dzSt * (fFSEzRe * cRe - fFSEzIm * cIm));
				EzIm -= (float)(dxFi_dzSt * (fFSEzRe * cIm + fFSEzIm * cRe));
			}
			if (dzFi != 0.f)
			{
				bRe = DataPtrs.ExpArrZFi[Two_iz], bIm = DataPtrs.ExpArrZFi[Two_iz_p_1];
				cRe = ExpXFiRe * bRe - ExpXFiIm * bIm; cIm = ExpXFiRe * bIm + ExpXFiIm * bRe;

				ExRe += (float)(dxFi_dzFi * (fFFExRe * cRe - fFFExIm * cIm));
				ExIm += (float)(dxFi_dzFi * (fFFExRe * cIm + fFFExIm * cRe));
				EzRe += (float)(dxFi_dzFi * (fFFEzRe * cRe - fFFEzIm * cIm));
				EzIm += (float)(dxFi_dzFi * (fFFEzRe * cIm + fFFEzIm * cRe));
			}
		}
		if (dzSt != 0.f)
		{
			float ExpZStRe = DataPtrs.ExpArrZSt[Two_iz], ExpZStIm = DataPtrs.ExpArrZSt[Two_iz_p_1];

			bRe = DataPtrs.FFTArrZStEx[Two_ix]; bIm = DataPtrs.FFTArrZStEx[Two_ix_p_1];
			ExRe += (float)(dzSt * (ExpZStRe * bRe - ExpZStIm * bIm));
			ExIm += (float)(dzSt * (ExpZStRe * bIm + ExpZStIm * bRe));

			bRe = DataPtrs.FFTArrZStEz[Two_ix]; bIm = DataPtrs.FFTArrZStEz[Two_ix_p_1];
			EzRe += (float)(DataPtrs.dzSt * (ExpZStRe * bRe - ExpZStIm * bIm));
			EzIm += (float)(DataPtrs.dzSt * (ExpZStRe * bIm + ExpZStIm * bRe));
		}
		if (dzFi != 0.f)
		{
			float ExpZFiRe = DataPtrs.ExpArrZFi[Two_iz], ExpZFiIm = DataPtrs.ExpArrZFi[Two_iz_p_1];

			bRe = DataPtrs.FFTArrZFiEx[Two_ix]; bIm = DataPtrs.FFTArrZFiEx[Two_ix_p_1];
			ExRe -= (float)(dzFi * (ExpZFiRe * bRe - ExpZFiIm * bIm));
			ExIm -= (float)(dzFi * (ExpZFiRe * bIm + ExpZFiIm * bRe));

			bRe = DataPtrs.FFTArrZFiEz[Two_ix]; bIm = DataPtrs.FFTArrZFiEz[Two_ix_p_1];
			EzRe -= (float)(dzFi * (ExpZFiRe * bRe - ExpZFiIm * bIm));
			EzIm -= (float)(dzFi * (ExpZFiRe * bIm + ExpZFiIm * bRe));
		}

		*tEx = ExRe; *(tEx + 1) = ExIm;
		*tEz = EzRe; *(tEz + 1) = EzIm;
    }
}

void srTGenOptElem::MakeWfrEdgeCorrection_GPU(srTSRWRadStructAccessData* RadAccessData, float* pDataEx, float* pDataEz, srTDataPtrsForWfrEdgeCorr& DataPtrs, TGPUUsageArg* pGPU)
{
	pDataEx = (float*)CAuxGPU::ToDevice(pGPU, pDataEx, 2*RadAccessData->ne*RadAccessData->nx*RadAccessData->nz*sizeof(float));
	pDataEz = (float*)CAuxGPU::ToDevice(pGPU, pDataEz, 2*RadAccessData->ne*RadAccessData->nx*RadAccessData->nz*sizeof(float));
	DataPtrs.FFTArrXStEx = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrXStEx, 2*RadAccessData->nz*sizeof(float));
	DataPtrs.FFTArrXStEz = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrXStEz, 2*RadAccessData->nz*sizeof(float));
	DataPtrs.FFTArrXFiEx = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrXFiEx, 2*RadAccessData->nz*sizeof(float));
	DataPtrs.FFTArrXFiEz = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrXFiEz, 2*RadAccessData->nz*sizeof(float));
	DataPtrs.FFTArrZStEx = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrZStEx, 2*RadAccessData->nx*sizeof(float));
	DataPtrs.FFTArrZStEz = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrZStEz, 2*RadAccessData->nx*sizeof(float));
	DataPtrs.FFTArrZFiEx = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrZFiEx, 2*RadAccessData->nx*sizeof(float));
	DataPtrs.FFTArrZFiEz = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.FFTArrZFiEz, 2*RadAccessData->nx*sizeof(float));
	DataPtrs.ExpArrXSt = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.ExpArrXSt, 2*RadAccessData->nx*sizeof(float));
	DataPtrs.ExpArrXFi = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.ExpArrXFi, 2*RadAccessData->nx*sizeof(float));
	DataPtrs.ExpArrZSt = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.ExpArrZSt, 2*RadAccessData->nz*sizeof(float));
	DataPtrs.ExpArrZFi = (float*)CAuxGPU::ToDevice(pGPU, DataPtrs.ExpArrZFi, 2*RadAccessData->nz*sizeof(float));

	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pDataEx);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pDataEz);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrXStEx);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrXStEz);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrXFiEx);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrXFiEz);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrZStEx);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrZStEz);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrZFiEx);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.FFTArrZFiEz);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.ExpArrXSt);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.ExpArrXFi);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.ExpArrZSt);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, DataPtrs.ExpArrZFi);

	srTSRWRadStructAccessData* pRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, RadAccessData, sizeof(srTSRWRadStructAccessData)); //HG27072024
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData_dev);

	const int bs = 256;
	dim3 blocks(RadAccessData->nx / bs + ((RadAccessData->nx & (bs - 1)) != 0), RadAccessData->nz);
	dim3 threads(bs, 1);
	//MakeWfrEdgeCorrection_Kernel << <blocks, threads >> > (*RadAccessData, pDataEx, pDataEz, DataPtrs, (float)DataPtrs.dxSt, (float)DataPtrs.dxFi, (float)DataPtrs.dzSt, (float)DataPtrs.dzFi);
	MakeWfrEdgeCorrection_Kernel << <blocks, threads >> > (pRadAccessData_dev, pDataEx, pDataEz, DataPtrs, (float)DataPtrs.dxSt, (float)DataPtrs.dxFi, (float)DataPtrs.dzSt, (float)DataPtrs.dzFi); //HG27072024

	CAuxGPU::ToHostAndFree(pGPU, pRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true); //HG27072024

	DataPtrs.FFTArrXStEx = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrXStEx, 2*RadAccessData->nz*sizeof(float), true);
	DataPtrs.FFTArrXStEz = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrXStEz, 2*RadAccessData->nz*sizeof(float), true);
	DataPtrs.FFTArrXFiEx = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrXFiEx, 2*RadAccessData->nz*sizeof(float), true);
	DataPtrs.FFTArrXFiEz = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrXFiEz, 2*RadAccessData->nz*sizeof(float), true);
	DataPtrs.FFTArrZStEx = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrZStEx, 2*RadAccessData->nx*sizeof(float), true);
	DataPtrs.FFTArrZStEz = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrZStEz, 2*RadAccessData->nx*sizeof(float), true);
	DataPtrs.FFTArrZFiEx = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrZFiEx, 2*RadAccessData->nx*sizeof(float), true);
	DataPtrs.FFTArrZFiEz = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.FFTArrZFiEz, 2*RadAccessData->nx*sizeof(float), true);
	DataPtrs.ExpArrXSt = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.ExpArrXSt, 2*RadAccessData->nx*sizeof(float), true);
	DataPtrs.ExpArrXFi = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.ExpArrXFi, 2*RadAccessData->nx*sizeof(float), true);
	DataPtrs.ExpArrZSt = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.ExpArrZSt, 2*RadAccessData->nz*sizeof(float), true);
	DataPtrs.ExpArrZFi = (float*)CAuxGPU::ToHostAndFree(pGPU, DataPtrs.ExpArrZFi, 2*RadAccessData->nz*sizeof(float), true);

	CAuxGPU::MarkUpdated(pGPU, pDataEx, true, false);
	CAuxGPU::MarkUpdated(pGPU, pDataEz, true, false);

//#ifdef _DEBUG
//	CAuxGPU::ToHostAndFree(pGPU, pDataEx, 2*RadAccessData->ne*RadAccessData->nx*RadAccessData->nz*sizeof(float));
//	CAuxGPU::ToHostAndFree(pGPU, pDataEz, 2*RadAccessData->ne*RadAccessData->nx*RadAccessData->nz*sizeof(float));
//	cudaStreamSynchronize(0);
//	auto err = cudaGetLastError();
//	printf("%s\r\n", cudaGetErrorString(err));
//#endif
}

template<bool TreatPolCompX, bool TreatPolCompZ> __global__ void RadResizeCore_Kernel(srTSRWRadStructAccessData* pOldRadAccessData, srTSRWRadStructAccessData* pNewRadAccessData)
{
	int ixStart = int(pNewRadAccessData->AuxLong1);
	int ixEnd = int(pNewRadAccessData->AuxLong2);
	int izStart = int(pNewRadAccessData->AuxLong3);
	int izEnd = int(pNewRadAccessData->AuxLong4);

    int ix = (blockIdx.x * blockDim.x + threadIdx.x) + ixStart; //nx range
    int iz = (blockIdx.y * blockDim.y + threadIdx.y) + izStart; //nz range
    int ie = (blockIdx.z * blockDim.z + threadIdx.z); //ne range

	if (ix > ixEnd) return;
	if (iz > izEnd) return;

	const double DistAbsTol = 1.E-10;
	double xStepInvOld = 1./pOldRadAccessData->xStep;
	double zStepInvOld = 1./pOldRadAccessData->zStep;
	int nx_mi_1Old = pOldRadAccessData->nx - 1;
	int nz_mi_1Old = pOldRadAccessData->nz - 1;
	int nx_mi_2Old = nx_mi_1Old - 1;
	int nz_mi_2Old = nz_mi_1Old - 1;

	//OC31102018: moved by SY at parallelizing SRW via OpenMP
	//srTInterpolAux01 InterpolAux01;
	//srTInterpolAux02 InterpolAux02[4], InterpolAux02I[2];
	//srTInterpolAuxF AuxF[4], AuxFI[2];
	//int ixStOld, izStOld, ixStOldPrev = -1000, izStOldPrev = -1000;

	//long PerX_New = pNewRadAccessData->ne << 1;
	//long PerZ_New = PerX_New*pNewRadAccessData->nx;
	long long PerX_New = pNewRadAccessData->ne << 1;
	long long PerZ_New = PerX_New*pNewRadAccessData->nx;

	//long PerX_Old = PerX_New;
	//long PerZ_Old = PerX_Old*pOldRadAccessData->nx;
	long long PerX_Old = PerX_New;
	long long PerZ_Old = PerX_Old*pOldRadAccessData->nx;

	float *pEX0_New = 0, *pEZ0_New = 0;
	pEX0_New = pNewRadAccessData->pBaseRadX;
	pEZ0_New = pNewRadAccessData->pBaseRadZ;

	float* pEX0_Old = 0, * pEZ0_Old = 0;
	pEX0_Old = pOldRadAccessData->pBaseRadX;
	pEZ0_Old = pOldRadAccessData->pBaseRadZ;

	
	int ixStOld, izStOld; //OC25012024 //ixStOldPrev = -1000, izStOldPrev = -1000;
	//int ixStOld, izStOld, ixStOldPrev = -1000, izStOldPrev = -1000;
	//SY: do we need this (always returns 0, updates some clock)
	//if(result = srYield.Check()) return result;

	double zAbs = pNewRadAccessData->zStart + iz * pNewRadAccessData->zStep;

	char FieldShouldBeZeroedDueToZ = 0;
	if (pNewRadAccessData->WfrEdgeCorrShouldBeDone)
	{
		if ((zAbs < pNewRadAccessData->zWfrMin - DistAbsTol) || (zAbs > pNewRadAccessData->zWfrMax + DistAbsTol)) FieldShouldBeZeroedDueToZ = 1;
	}

	int izcOld = int((zAbs - pOldRadAccessData->zStart) * zStepInvOld + 1.E-06);

	double zRel = zAbs - (pOldRadAccessData->zStart + izcOld * pOldRadAccessData->zStep);

	if (izcOld == nz_mi_1Old) { izStOld = izcOld - 3; zRel += 2. * pOldRadAccessData->zStep; }
	else if (izcOld == nz_mi_2Old) { izStOld = izcOld - 2; zRel += pOldRadAccessData->zStep; }
	else if (izcOld == 0) { izStOld = izcOld; zRel -= pOldRadAccessData->zStep; }
	else izStOld = izcOld - 1;

	zRel *= zStepInvOld;

	int izcOld_mi_izStOld = izcOld - izStOld;
	//long izPerZ_New = iz*PerZ_New;
	long long izPerZ_New = iz * PerZ_New;

	double xAbs = pNewRadAccessData->xStart + ix * pNewRadAccessData->xStep;

	char FieldShouldBeZeroedDueToX = 0;
	if (pNewRadAccessData->WfrEdgeCorrShouldBeDone)
	{
		if ((xAbs < pNewRadAccessData->xWfrMin - DistAbsTol) || (xAbs > pNewRadAccessData->xWfrMax + DistAbsTol)) FieldShouldBeZeroedDueToX = 1;
	}
	char FieldShouldBeZeroed = (FieldShouldBeZeroedDueToX || FieldShouldBeZeroedDueToZ);

	int ixcOld = int((xAbs - pOldRadAccessData->xStart) * xStepInvOld + 1.E-06);
	double xRel = xAbs - (pOldRadAccessData->xStart + ixcOld * pOldRadAccessData->xStep);

	if (ixcOld == nx_mi_1Old) { ixStOld = ixcOld - 3; xRel += 2. * pOldRadAccessData->xStep; }
	else if (ixcOld == nx_mi_2Old) { ixStOld = ixcOld - 2; xRel += pOldRadAccessData->xStep; }
	else if (ixcOld == 0) { ixStOld = ixcOld; xRel -= pOldRadAccessData->xStep; }
	else ixStOld = ixcOld - 1;

	xRel *= xStepInvOld;

	int ixcOld_mi_ixStOld = ixcOld - ixStOld;

	//or (int ie = 0; ie < pNewRadAccessData->ne; ie++)
	{
		//OC31102018: modified by SY at OpenMP parallelization
		//ixStOldPrev = -1000; izStOldPrev = -1000;

		//OC31102018: moved by SY at OpenMP parallelization
		srTInterpolAux01 InterpolAux01;
		srTInterpolAux02 InterpolAux02[4], InterpolAux02I[2];
		srTInterpolAuxF AuxF[4], AuxFI[2];
		//ixStOldPrev = -1000; izStOldPrev = -1000; //OC25012024 (commented-out: never used?)
		float BufF[4], BufFI[2];
		char UseLowOrderInterp_PolCompX = 0, UseLowOrderInterp_PolCompZ = 0;

		//long Two_ie = ie << 1;
		long long Two_ie = ie << 1;

		float* pEX_StartForX_New = 0, * pEZ_StartForX_New = 0;
		pEX_StartForX_New = pEX0_New + izPerZ_New;
		pEZ_StartForX_New = pEZ0_New + izPerZ_New;

		//long ixPerX_New_p_Two_ie = ix*PerX_New + Two_ie;
		long long ixPerX_New_p_Two_ie = ix * PerX_New + Two_ie;
		float* pEX_New = 0, * pEZ_New = 0;
		pEX_New = pEX_StartForX_New + ixPerX_New_p_Two_ie;
		pEZ_New = pEZ_StartForX_New + ixPerX_New_p_Two_ie;

		//long TotOffsetOld = izStOld*PerZ_Old + ixStOld*PerX_Old + Two_ie;
		long long TotOffsetOld = izStOld * PerZ_Old + ixStOld * PerX_Old + Two_ie;

		if (TreatPolCompX)
		{
			float* pExSt_Old = pEX0_Old + TotOffsetOld;
			srTGenOptElem::GetCellDataForInterpol(pExSt_Old, PerX_Old, PerZ_Old, AuxF);

			srTGenOptElem::SetupCellDataI(AuxF, AuxFI);
			UseLowOrderInterp_PolCompX = srTGenOptElem::CheckForLowOrderInterp(AuxF, AuxFI, ixcOld_mi_ixStOld, izcOld_mi_izStOld, &InterpolAux01, InterpolAux02, InterpolAux02I);

			if (!UseLowOrderInterp_PolCompX)
			{
				for (int i = 0; i < 2; i++)
				{
					srTGenOptElem::SetupInterpolAux02(AuxF + i, &InterpolAux01, InterpolAux02 + i);
				}
				srTGenOptElem::SetupInterpolAux02(AuxFI, &InterpolAux01, InterpolAux02I);
			}

			if (UseLowOrderInterp_PolCompX)
			{
				srTGenOptElem::InterpolF_LowOrder(InterpolAux02, xRel, zRel, BufF, 0);
				srTGenOptElem::InterpolFI_LowOrder(InterpolAux02I, xRel, zRel, BufFI, 0);
			}
			else
			{
				srTGenOptElem::InterpolF(InterpolAux02, xRel, zRel, BufF, 0);
				srTGenOptElem::InterpolFI(InterpolAux02I, xRel, zRel, BufFI, 0);
			}

			(*BufFI) *= AuxFI->fNorm;
			srTGenOptElem::ImproveReAndIm(BufF, BufFI);

			if (FieldShouldBeZeroed)
			{
				*BufF = 0.; *(BufF + 1) = 0.;
			}

			*pEX_New = *BufF;
			*(pEX_New + 1) = *(BufF + 1);
		}
		if (TreatPolCompZ)
		{
			float* pEzSt_Old = pEZ0_Old + TotOffsetOld;
			srTGenOptElem::GetCellDataForInterpol(pEzSt_Old, PerX_Old, PerZ_Old, AuxF + 2);

			srTGenOptElem::SetupCellDataI(AuxF + 2, AuxFI + 1);
			UseLowOrderInterp_PolCompZ = srTGenOptElem::CheckForLowOrderInterp(AuxF + 2, AuxFI + 1, ixcOld_mi_ixStOld, izcOld_mi_izStOld, &InterpolAux01, InterpolAux02 + 2, InterpolAux02I + 1);

			if (!UseLowOrderInterp_PolCompZ)
			{
				for (int i = 0; i < 2; i++)
				{
					srTGenOptElem::SetupInterpolAux02(AuxF + 2 + i, &InterpolAux01, InterpolAux02 + 2 + i);
				}
				srTGenOptElem::SetupInterpolAux02(AuxFI + 1, &InterpolAux01, InterpolAux02I + 1);
			}
			
			if (UseLowOrderInterp_PolCompZ)
			{
				srTGenOptElem::InterpolF_LowOrder(InterpolAux02, xRel, zRel, BufF, 2);
				srTGenOptElem::InterpolFI_LowOrder(InterpolAux02I, xRel, zRel, BufFI, 1);
			}
			else
			{
				srTGenOptElem::InterpolF(InterpolAux02, xRel, zRel, BufF, 2);
				srTGenOptElem::InterpolFI(InterpolAux02I, xRel, zRel, BufFI, 1);
			}

			(*(BufFI + 1)) *= (AuxFI + 1)->fNorm;
			srTGenOptElem::ImproveReAndIm(BufF + 2, BufFI + 1);

			if (FieldShouldBeZeroed)
			{
				*(BufF + 2) = 0.; *(BufF + 3) = 0.;
			}

			*pEZ_New = *(BufF + 2);
			*(pEZ_New + 1) = *(BufF + 3);
		}
	}
}

int srTGenOptElem::RadResizeCore_GPU(srTSRWRadStructAccessData& OldRadAccessData, srTSRWRadStructAccessData& NewRadAccessData, char PolComp, TGPUUsageArg* pGPU)
{
	char TreatPolCompX = ((PolComp == 0) || (PolComp == 'x'));
	char TreatPolCompZ = ((PolComp == 0) || (PolComp == 'z'));

	int nx = NewRadAccessData.AuxLong2 - NewRadAccessData.AuxLong1 + 1;
	int nz = NewRadAccessData.AuxLong4 - NewRadAccessData.AuxLong3 + 1;
	int ne = NewRadAccessData.ne;
	OldRadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, OldRadAccessData.pBaseRadX, 2*OldRadAccessData.ne*OldRadAccessData.nx*OldRadAccessData.nz*sizeof(float));
	OldRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, OldRadAccessData.pBaseRadZ, 2*OldRadAccessData.ne*OldRadAccessData.nx*OldRadAccessData.nz*sizeof(float));
	NewRadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, NewRadAccessData.pBaseRadX, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float), true, false, 0);
	NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, NewRadAccessData.pBaseRadZ, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float), true, false, 0);
	
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, OldRadAccessData.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, OldRadAccessData.pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, NewRadAccessData.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, NewRadAccessData.pBaseRadZ);

	srTSRWRadStructAccessData* pOldRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &OldRadAccessData, sizeof(srTSRWRadStructAccessData)); //HG27072024
	srTSRWRadStructAccessData* pNewRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &NewRadAccessData, sizeof(srTSRWRadStructAccessData)); //HG27072024
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pOldRadAccessData_dev); //HG27072024
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pNewRadAccessData_dev); //HG27072024

	const int bs = 32;
	dim3 blocks(nx / bs + ((nx & (bs - 1)) != 0), nz, ne);
	dim3 threads(bs, 1);
	
	if (TreatPolCompX && TreatPolCompZ) RadResizeCore_Kernel<true, true> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);
	else if (TreatPolCompX) RadResizeCore_Kernel<true, false> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);
	else if (TreatPolCompZ) RadResizeCore_Kernel<false, true> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);

	CAuxGPU::ToHostAndFree(pGPU, pOldRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true); //HG27072024
	CAuxGPU::ToHostAndFree(pGPU, pNewRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true); //HG27072024

	OldRadAccessData.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, OldRadAccessData.pBaseRadX, 2*OldRadAccessData.ne*OldRadAccessData.nx*OldRadAccessData.nz*sizeof(float), true);
	OldRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, OldRadAccessData.pBaseRadZ, 2*OldRadAccessData.ne*OldRadAccessData.nx*OldRadAccessData.nz*sizeof(float), true);
	//NewRadAccessData.pBaseRadX = CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadX, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float));
	//NewRadAccessData.pBaseRadZ = CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadZ, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float));
	CAuxGPU::MarkUpdated(pGPU, NewRadAccessData.pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, NewRadAccessData.pBaseRadZ, true, false);
//#ifndef _DEBUG
	NewRadAccessData.pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, NewRadAccessData.pBaseRadX);
	NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, NewRadAccessData.pBaseRadZ);
//#endif

//#ifdef _DEBUG
	//NewRadAccessData.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadX, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float), false);
	//NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadZ, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float), false);
	//cudaStreamSynchronize(0);
	//auto err = cudaGetLastError();
	//printf("%s\r\n", cudaGetErrorString(err));

//#endif

	return 0;
}

template<bool TreatPolCompX, bool TreatPolCompZ> __global__ void RadResizeCore_OnlyLargerRange_Kernel(srTSRWRadStructAccessData* pOldRadAccessData, srTSRWRadStructAccessData* pNewRadAccessData)
{

	int ixStart = int(pNewRadAccessData->AuxLong1);
	int ixEnd = int(pNewRadAccessData->AuxLong2);
	int izStart = int(pNewRadAccessData->AuxLong3);
	int izEnd = int(pNewRadAccessData->AuxLong4);


	int ix = (blockIdx.x * blockDim.x + threadIdx.x) + ixStart; //nx range
	int iz = (blockIdx.y * blockDim.y + threadIdx.y) + izStart; //nz range
	int ie = (blockIdx.z * blockDim.z + threadIdx.z); //ne range

	if (ix > ixEnd) return;
	if (iz > izEnd) return;

	float* pEX0_New = pNewRadAccessData->pBaseRadX;
	float* pEZ0_New = pNewRadAccessData->pBaseRadZ;

	float* pEX0_Old = pOldRadAccessData->pBaseRadX;
	float* pEZ0_Old = pOldRadAccessData->pBaseRadZ;

	//long PerX_New = pNewRadAccessData->ne << 1;
	//long PerZ_New = PerX_New*pNewRadAccessData->nx;
	long long PerX_New = pNewRadAccessData->ne << 1;
	long long PerZ_New = PerX_New*pNewRadAccessData->nx;

	//long PerX_Old = PerX_New;
	//long PerZ_Old = PerX_Old*pOldRadAccessData->nx;
	long long PerX_Old = PerX_New;
	long long PerZ_Old = PerX_Old*pOldRadAccessData->nx;

	double xStepInvOld = 1./pOldRadAccessData->xStep;
	double zStepInvOld = 1./pOldRadAccessData->zStep;
	
	//long Two_ie = ie << 1;
	long long Two_ie = ie << 1;
	
	//long izPerZ_New = iz*PerZ_New;
	long long izPerZ_New = iz*PerZ_New;
	float* pEX_StartForX_New = pEX0_New + izPerZ_New;
	float* pEZ_StartForX_New = pEZ0_New + izPerZ_New;

	//long izPerZ_Old = (iz - izStart)*PerZ_Old;

	double zAbs = pNewRadAccessData->zStart + iz*pNewRadAccessData->zStep;
	long izOld = long((zAbs - pOldRadAccessData->zStart)*zStepInvOld + 1.E-08);
	//long izPerZ_Old = izOld*PerZ_Old;
	long long izPerZ_Old = izOld*PerZ_Old;

	float* pEX_StartForX_Old = pEX0_Old + izPerZ_Old;
	float* pEZ_StartForX_Old = pEZ0_Old + izPerZ_Old;

	//long ixPerX_New_p_Two_ie = ix*PerX_New + Two_ie;
	long long ixPerX_New_p_Two_ie = ix*PerX_New + Two_ie;
	float* pEX_New = pEX_StartForX_New + ixPerX_New_p_Two_ie;
	float* pEZ_New = pEZ_StartForX_New + ixPerX_New_p_Two_ie;

	//long ixPerX_Old_p_Two_ie = (ix - ixStart)*PerX_Old + Two_ie;

	double xAbs = pNewRadAccessData->xStart + ix*pNewRadAccessData->xStep;
	long ixOld = long((xAbs - pOldRadAccessData->xStart)*xStepInvOld + 1.E-08);
	//long ixPerX_Old_p_Two_ie = ixOld*PerX_Old + Two_ie;
	long long ixPerX_Old_p_Two_ie = ixOld*PerX_Old + Two_ie;

	float* pEX_Old = pEX_StartForX_Old + ixPerX_Old_p_Two_ie;
	float* pEZ_Old = pEZ_StartForX_Old + ixPerX_Old_p_Two_ie;

	if (TreatPolCompX) { *pEX_New = *pEX_Old; *(pEX_New + 1) = *(pEX_Old + 1); }
	if (TreatPolCompZ) { *pEZ_New = *pEZ_Old; *(pEZ_New + 1) = *(pEZ_Old + 1); }
}


int srTGenOptElem::RadResizeCore_OnlyLargerRange_GPU(srTSRWRadStructAccessData& OldRadAccessData, srTSRWRadStructAccessData& NewRadAccessData, char PolComp, TGPUUsageArg* pGPU)
{
	char TreatPolCompX = ((PolComp == 0) || (PolComp == 'x'));
	char TreatPolCompZ = ((PolComp == 0) || (PolComp == 'z'));

	int nx = NewRadAccessData.AuxLong2 - NewRadAccessData.AuxLong1 + 1;
	int nz = NewRadAccessData.AuxLong4 - NewRadAccessData.AuxLong3 + 1;
	int ne = NewRadAccessData.ne;

	OldRadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, OldRadAccessData.pBaseRadX, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float));
	OldRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, OldRadAccessData.pBaseRadZ, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float));
	NewRadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, NewRadAccessData.pBaseRadX, 2 * NewRadAccessData.ne * NewRadAccessData.nx * NewRadAccessData.nz * sizeof(float), true, false, 1);
	NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, NewRadAccessData.pBaseRadZ, 2 * NewRadAccessData.ne * NewRadAccessData.nx * NewRadAccessData.nz * sizeof(float), true, false, 1);

	CAuxGPU::EnsureDeviceMemoryReady(pGPU, OldRadAccessData.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, OldRadAccessData.pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, NewRadAccessData.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, NewRadAccessData.pBaseRadZ);

	srTSRWRadStructAccessData* pOldRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &OldRadAccessData, sizeof(srTSRWRadStructAccessData));
	srTSRWRadStructAccessData* pNewRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &NewRadAccessData, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pOldRadAccessData_dev);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pNewRadAccessData_dev);

	const int bs = 32;
	dim3 blocks(nx / bs + ((nx & (bs - 1)) != 0), nz, ne);
	dim3 threads(bs, 1);

	if (TreatPolCompX && TreatPolCompZ) RadResizeCore_OnlyLargerRange_Kernel<true, true> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);
	else if (TreatPolCompX) RadResizeCore_OnlyLargerRange_Kernel<true, false> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);
	else if (TreatPolCompZ) RadResizeCore_OnlyLargerRange_Kernel<false, true> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);

	CAuxGPU::ToHostAndFree(pGPU, pOldRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true);
	CAuxGPU::ToHostAndFree(pGPU, pNewRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true);

	OldRadAccessData.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, OldRadAccessData.pBaseRadX, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float), true);
	OldRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, OldRadAccessData.pBaseRadZ, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float), true);
	
	CAuxGPU::MarkUpdated(pGPU, NewRadAccessData.pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, NewRadAccessData.pBaseRadZ, true, false);
	
	NewRadAccessData.pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, NewRadAccessData.pBaseRadX);
	NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, NewRadAccessData.pBaseRadZ);
	
	return 0;
}

template<bool TreatPolCompX, bool TreatPolCompZ> __global__ void RadResizeCore_OnlyLargerRangeE_Kernel(srTSRWRadStructAccessData* pOldRadAccessData, srTSRWRadStructAccessData* pNewRadAccessData)
{
	int ieStart = int(pNewRadAccessData->AuxLong1);
	
	int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
	int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range
	int ie = (blockIdx.z * blockDim.z + threadIdx.z) + ieStart; //ne range

	if (ix > pNewRadAccessData->nx) return;
	if (iz > pNewRadAccessData->nz) return;
	
	float* pEX0_New = pNewRadAccessData->pBaseRadX;
	float* pEZ0_New = pNewRadAccessData->pBaseRadZ;

	float* pEX0_Old = pOldRadAccessData->pBaseRadX;
	float* pEZ0_Old = pOldRadAccessData->pBaseRadZ;

	//long PerX_New = pNewRadAccessData->ne << 1;
	//long PerZ_New = PerX_New*pNewRadAccessData->nx;
	long long PerX_New = pNewRadAccessData->ne << 1;
	long long PerZ_New = PerX_New * pNewRadAccessData->nx;

	//long PerX_Old = pOldRadAccessData->ne << 1;
	//long PerZ_Old = PerX_Old*pOldRadAccessData->nx;
	long long PerX_Old = pOldRadAccessData->ne << 1;
	long long PerZ_Old = PerX_Old * pOldRadAccessData->nx;

	double eStepInvOld = 1. / pOldRadAccessData->eStep;

	//long iz_PerZ_New = iz*PerZ_New;
	//long iz_PerZ_Old = iz*PerZ_Old;
	long long iz_PerZ_New = iz * PerZ_New;
	long long iz_PerZ_Old = iz * PerZ_Old;

	//long iz_PerZ_New_p_ix_PerX_New = iz_PerZ_New + ix*PerX_New;
	//long iz_PerZ_Old_p_ix_PerX_Old = iz_PerZ_Old + ix*PerX_Old;
	long long iz_PerZ_New_p_ix_PerX_New = iz_PerZ_New + ix * PerX_New;
	long long iz_PerZ_Old_p_ix_PerX_Old = iz_PerZ_Old + ix * PerX_Old;

	//long ofstNew = iz_PerZ_New_p_ix_PerX_New + (ie << 1);
	long long ofstNew = iz_PerZ_New_p_ix_PerX_New + (ie << 1);
	float* pEX_New = pEX0_New + ofstNew;
	float* pEZ_New = pEZ0_New + ofstNew;

	double eAbs = pNewRadAccessData->eStart + ie * pNewRadAccessData->eStep;
	long ieOld = long((eAbs - pOldRadAccessData->eStart) * eStepInvOld + 1.E-08);

	//long ofstOld = iz_PerZ_Old_p_ix_PerX_Old + (ieOld << 1);
	long long ofstOld = iz_PerZ_Old_p_ix_PerX_Old + (ieOld << 1);
	float* pEX_Old = pEX0_Old + ofstOld;
	float* pEZ_Old = pEZ0_Old + ofstOld;

	if (TreatPolCompX) { *pEX_New = *pEX_Old; *(pEX_New + 1) = *(pEX_Old + 1); }
	if (TreatPolCompZ) { *pEZ_New = *pEZ_Old; *(pEZ_New + 1) = *(pEZ_Old + 1); }
}


int srTGenOptElem::RadResizeCore_OnlyLargerRangeE_GPU(srTSRWRadStructAccessData& OldRadAccessData, srTSRWRadStructAccessData& NewRadAccessData, char PolComp, TGPUUsageArg* pGPU)
{
	char TreatPolCompX = ((PolComp == 0) || (PolComp == 'x')) && (OldRadAccessData.pBaseRadX != 0);
	char TreatPolCompZ = ((PolComp == 0) || (PolComp == 'z')) && (OldRadAccessData.pBaseRadZ != 0);

	int ne = NewRadAccessData.AuxLong2 - NewRadAccessData.AuxLong1 + 1;

	OldRadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, OldRadAccessData.pBaseRadX, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float));
	OldRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, OldRadAccessData.pBaseRadZ, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float));
	NewRadAccessData.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, NewRadAccessData.pBaseRadX, 2 * NewRadAccessData.ne * NewRadAccessData.nx * NewRadAccessData.nz * sizeof(float), true);
	NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, NewRadAccessData.pBaseRadZ, 2 * NewRadAccessData.ne * NewRadAccessData.nx * NewRadAccessData.nz * sizeof(float), true);

	CAuxGPU::EnsureDeviceMemoryReady(pGPU, OldRadAccessData.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, OldRadAccessData.pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, NewRadAccessData.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, NewRadAccessData.pBaseRadZ);

	srTSRWRadStructAccessData* pOldRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &OldRadAccessData, sizeof(srTSRWRadStructAccessData)); //HG27072024
	srTSRWRadStructAccessData* pNewRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &NewRadAccessData, sizeof(srTSRWRadStructAccessData)); //HG27072024
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pOldRadAccessData_dev);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pNewRadAccessData_dev);

	const int bs = 32;
	dim3 blocks(NewRadAccessData.nx / bs + ((NewRadAccessData.nx & (bs - 1)) != 0), NewRadAccessData.nz, ne);
	dim3 threads(bs, 1);

	if (TreatPolCompX && TreatPolCompZ) RadResizeCore_OnlyLargerRangeE_Kernel<true, true> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);
	else if (TreatPolCompX) RadResizeCore_OnlyLargerRangeE_Kernel<true, false> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);
	else if (TreatPolCompZ) RadResizeCore_OnlyLargerRangeE_Kernel<false, true> << <blocks, threads >> > (pOldRadAccessData_dev, pNewRadAccessData_dev);

	CAuxGPU::ToHostAndFree(pGPU, pOldRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true); //HG27072024
	CAuxGPU::ToHostAndFree(pGPU, pNewRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true); //HG27072024

	OldRadAccessData.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, OldRadAccessData.pBaseRadX, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float), true);
	OldRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, OldRadAccessData.pBaseRadZ, 2 * OldRadAccessData.ne * OldRadAccessData.nx * OldRadAccessData.nz * sizeof(float), true);
	//NewRadAccessData.pBaseRadX = CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadX, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float));
	//NewRadAccessData.pBaseRadZ = CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadZ, 2*NewRadAccessData.ne*NewRadAccessData.nx*NewRadAccessData.nz*sizeof(float));
	CAuxGPU::MarkUpdated(pGPU, NewRadAccessData.pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, NewRadAccessData.pBaseRadZ, true, false);
	//#ifndef _DEBUG
	NewRadAccessData.pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, NewRadAccessData.pBaseRadX);
	NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, NewRadAccessData.pBaseRadZ);
	//#endif

	//#ifdef _DEBUG
	//NewRadAccessData.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadX, 2 * NewRadAccessData.ne * NewRadAccessData.nx * NewRadAccessData.nz * sizeof(float), false);
	//NewRadAccessData.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, NewRadAccessData.pBaseRadZ, 2 * NewRadAccessData.ne * NewRadAccessData.nx * NewRadAccessData.nz * sizeof(float), false);
	//cudaStreamSynchronize(0);
	//auto err = cudaGetLastError();
	//printf("%s\r\n", cudaGetErrorString(err));

	//#endif

	return 0;
}

__global__ void ExtractRadSliceConstE_Kernel(srTSRWRadStructAccessData *pRadAccessData, long ie, float* pOutEx, float* pOutEz)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
	int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range
	
	if(ix >= pRadAccessData->nx) return;
	if(iz >= pRadAccessData->nz) return;

	float *pEx0 = pRadAccessData->pBaseRadX;
	float *pEz0 = pRadAccessData->pBaseRadZ;
	long long  PerX = pRadAccessData->ne << 1;
	long long PerZ = PerX*pRadAccessData->nx;
	long long izPerZ = iz * PerZ;
	long long ixPerX = ix * PerX;
	long long iePerE = ie << 1;
	long long ixPerX_p_iePerE = ixPerX + iePerE;
	
	float *tOutEx = pOutEx, *tOutEz = pOutEz;
	float *pEx = pEx0 + izPerZ + ixPerX_p_iePerE;
	float *pEz = pEz0 + izPerZ + ixPerX_p_iePerE;

	tOutEx += (iz*pRadAccessData->nx + ix) << 1;
	*tOutEx = *pEx; *(tOutEx + 1) = *(pEx + 1);

	tOutEz += (iz*pRadAccessData->nx + ix) << 1;
	*tOutEz = *pEz; *(tOutEz + 1) = *(pEz + 1);
}

int srTGenOptElem::ExtractRadSliceConstE_GPU(srTSRWRadStructAccessData* pRadAccessData, long ie, float* pOutEx, float* pOutEz, TGPUUsageArg* pGPU)
{
	//printf("ExtractRadSliceConstE_GPU\r\n Data Size: %llu\r\n", 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	//printf("Dst Size: %llu\r\n", 2 * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pRadAccessData->pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, pRadAccessData->pBaseRadX, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pRadAccessData->pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, pRadAccessData->pBaseRadZ, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pOutEx = (float*)CAuxGPU::ToDevice(pGPU, pOutEx, 2 * pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);
	pOutEz = (float*)CAuxGPU::ToDevice(pGPU, pOutEz, 2 * pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);
	
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData->pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData->pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pOutEx);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pOutEz);
	
	srTSRWRadStructAccessData* pRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, pRadAccessData, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData_dev);

	const int bs = 256;
	dim3 blocks(pRadAccessData->nx / bs + ((pRadAccessData->nx & (bs - 1)) != 0), pRadAccessData->nz, 1);
	dim3 threads(bs, 1);

	ExtractRadSliceConstE_Kernel <<<blocks, threads >>> (pRadAccessData_dev, ie, pOutEx, pOutEz);

	CAuxGPU::ToHostAndFree(pGPU, pRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true);
	
	CAuxGPU::MarkUpdated(pGPU, pOutEx, true, false);
	CAuxGPU::MarkUpdated(pGPU, pOutEz, true, false);
	pRadAccessData->pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, pRadAccessData->pBaseRadX);
	pRadAccessData->pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, pRadAccessData->pBaseRadZ);
	pOutEx = (float*)CAuxGPU::GetHostPtr(pGPU, pOutEx);
	pOutEz = (float*)CAuxGPU::GetHostPtr(pGPU, pOutEz);

	//pRadAccessData->pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, pRadAccessData->pBaseRadX, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);
	//pRadAccessData->pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, pRadAccessData->pBaseRadZ, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);
	return 0;
}

__global__ void UpdateGenRadStructSliceConstE_Meth_0_Kernel(srTSRWRadStructAccessData* pRadDataSliceConstE, int ie, srTSRWRadStructAccessData* pRadAccessData)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
	int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range
	
	int neCom = pRadAccessData->ne;
	int nxCom = pRadAccessData->nx;
	int nzCom = pRadAccessData->nz;

	if(ix >= nxCom) return;
	if(iz >= nzCom) return;
	
	float *pEx0 = pRadAccessData->pBaseRadX;
	float *pEz0 = pRadAccessData->pBaseRadZ;

	long long PerX = neCom << 1;
	long long PerZ = PerX*nxCom;
	long long iePerE = ie << 1;

	float *tSliceEx = pRadDataSliceConstE->pBaseRadX + iz*nxCom*2 + ix*2;
	float *tSliceEz = pRadDataSliceConstE->pBaseRadZ + iz*nxCom*2 + ix*2;

	float *pEx = pEx0 + iz*PerZ + ix*PerX + iePerE;
	float *pEz = pEz0 + iz*PerZ + ix*PerX + iePerE;

	*(pEx++) = *(tSliceEx++); *pEx = *(tSliceEx);
	*(pEz++) = *(tSliceEz++); *pEz = *(tSliceEz);
}

int srTGenOptElem::UpdateGenRadStructSliceConstE_Meth_0_GPU(srTSRWRadStructAccessData* pRadDataSliceConstE, int ie, srTSRWRadStructAccessData* pRadAccessData, TGPUUsageArg* pGPU)
{
	pRadAccessData->pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, pRadAccessData->pBaseRadX, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pRadAccessData->pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, pRadAccessData->pBaseRadZ, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pRadDataSliceConstE->pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, pRadDataSliceConstE->pBaseRadX, 2 * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pRadDataSliceConstE->pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, pRadDataSliceConstE->pBaseRadZ, 2 * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData->pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData->pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadDataSliceConstE->pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadDataSliceConstE->pBaseRadZ);
	
	srTSRWRadStructAccessData* pRadDataSliceConstE_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, pRadDataSliceConstE, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadDataSliceConstE_dev);

	srTSRWRadStructAccessData* pRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, pRadAccessData, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData_dev);

	const int bs = 256;
	dim3 blocks(pRadAccessData->nx / bs + ((pRadAccessData->nx & (bs - 1)) != 0), pRadAccessData->nz, 1);
	dim3 threads(bs, 1);

	UpdateGenRadStructSliceConstE_Meth_0_Kernel <<<blocks, threads >>> (pRadDataSliceConstE_dev, ie, pRadAccessData_dev);

	CAuxGPU::ToHostAndFree(pGPU, pRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true);
	CAuxGPU::ToHostAndFree(pGPU, pRadDataSliceConstE_dev, sizeof(srTSRWRadStructAccessData), true);
	
	pRadDataSliceConstE->pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, pRadDataSliceConstE->pBaseRadX, 2 * pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);
	pRadDataSliceConstE->pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, pRadDataSliceConstE->pBaseRadZ, 2 * pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);

	CAuxGPU::MarkUpdated(pGPU, pRadAccessData->pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, pRadAccessData->pBaseRadZ, true, false);
	//pRadAccessData->pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, pRadAccessData->pBaseRadX);
	//pRadAccessData->pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, pRadAccessData->pBaseRadZ);
	
	//pRadAccessData->pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, pRadAccessData->pBaseRadX, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	//pRadAccessData->pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, pRadAccessData->pBaseRadZ, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	
	return 0;
}

__global__ void ReInterpolateWfrSliceSingleE_Kernel(srTSRWRadStructAccessData *pOldRadSingleE, srTSRWRadStructAccessData *pNewRadMultiE, int ie)
{
	const double DistAbsTol = 1.E-10;
	bool TreatPolCompX=true, TreatPolCompZ=true;

	int ixStart = 0; //int(NewRadAccessData.AuxLong1);
	int ixEnd = pNewRadMultiE->nx - 1; //int(NewRadAccessData.AuxLong2);
	int izStart = 0; //int(NewRadAccessData.AuxLong3);
	int izEnd = pNewRadMultiE->nz - 1; //int(NewRadAccessData.AuxLong4);

	int ix = (blockIdx.x * blockDim.x + threadIdx.x) + ixStart; //nx range
	int iz = (blockIdx.y * blockDim.y + threadIdx.y) + izStart; //nz range

	if(ix > ixEnd) return;
	if(iz > izEnd) return;

	double xStepInvOld = 1./pOldRadSingleE->xStep;
	double zStepInvOld = 1./pOldRadSingleE->zStep;
	int nx_mi_1Old = pOldRadSingleE->nx - 1;
	int nz_mi_1Old = pOldRadSingleE->nz - 1;
	int nx_mi_2Old = nx_mi_1Old - 1;
	int nz_mi_2Old = nz_mi_1Old - 1;

	srTInterpolAux01 InterpolAux01;
	srTInterpolAux02 InterpolAux02[4], InterpolAux02I[2];
	srTInterpolAuxF AuxF[4], AuxFI[2];
	int ixStOld, izStOld, ixStOldPrev = -1000, izStOldPrev = -1000;

	float *pEX0_New = 0, *pEZ0_New = 0;
	if(TreatPolCompX) pEX0_New = pNewRadMultiE->pBaseRadX;
	if(TreatPolCompZ) pEZ0_New = pNewRadMultiE->pBaseRadZ;

	//long PerX_New = pNewRadMultiE->ne << 1;
	//long PerZ_New = PerX_New*pNewRadMultiE->nx;
	long long PerX_New = pNewRadMultiE->ne << 1;
	long long PerZ_New = PerX_New*pNewRadMultiE->nx;

	//long PerX_Old = 2; //PerX_New;
	//long PerZ_Old = PerX_Old*pOldRadSingleE->nx;
	long long PerX_Old = 2; //PerX_New;
	long long PerZ_Old = PerX_Old*pOldRadSingleE->nx;

	float BufF[4], BufFI[2];
	int UseLowOrderInterp_PolCompX, UseLowOrderInterp_PolCompZ;
	int result = 0;

	//for(int ie=0; ie<NewRadAccessData.ne; ie++)
	//{
	//ixStOldPrev = -1000; izStOldPrev = -1000;

	//long Two_ie = ie << 1;
	long long Two_ie = ie << 1;

	double zAbs = pNewRadMultiE->zStart + iz*pNewRadMultiE->zStep;
	char FieldShouldBeZeroedDueToZ = 0;
	if(pNewRadMultiE->WfrEdgeCorrShouldBeDone)
	{
		if((zAbs < pNewRadMultiE->zWfrMin - DistAbsTol) || (zAbs > pNewRadMultiE->zWfrMax + DistAbsTol)) FieldShouldBeZeroedDueToZ = 1;
	}
	int izcOld = int((zAbs - pOldRadSingleE->zStart)*zStepInvOld + 1.E-06);
	if((izcOld < 0) || (izcOld > nz_mi_1Old))
	{
		//set El. field to 0 for all ix
		FieldShouldBeZeroedDueToZ = 1;
	}

	double zRel = zAbs - (pOldRadSingleE->zStart + izcOld*pOldRadSingleE->zStep);

	if(izcOld == nz_mi_1Old) { izStOld = izcOld - 3; zRel += 2.*pOldRadSingleE->zStep;}
	else if(izcOld == nz_mi_2Old) { izStOld = izcOld - 2; zRel += pOldRadSingleE->zStep;}
	else if(izcOld == 0) { izStOld = izcOld; zRel -= pOldRadSingleE->zStep;}
	else izStOld = izcOld - 1;

	zRel *= zStepInvOld;
	int izcOld_mi_izStOld = izcOld - izStOld;
	//long izPerZ_New = iz*PerZ_New;
	long long izPerZ_New = iz*PerZ_New;

	float *pEX_StartForX_New = 0, *pEZ_StartForX_New = 0;
	if(TreatPolCompX) pEX_StartForX_New = pEX0_New + izPerZ_New;
	if(TreatPolCompZ) pEZ_StartForX_New = pEZ0_New + izPerZ_New;

	//long ixPerX_New_p_Two_ie = ix*PerX_New + Two_ie;
	long long ixPerX_New_p_Two_ie = ix*PerX_New + Two_ie;
	float *pEX_New = 0, *pEZ_New = 0;
	if(TreatPolCompX) pEX_New = pEX_StartForX_New + ixPerX_New_p_Two_ie;
	if(TreatPolCompZ) pEZ_New = pEZ_StartForX_New + ixPerX_New_p_Two_ie;

	double xAbs = pNewRadMultiE->xStart + ix*pNewRadMultiE->xStep;
	char FieldShouldBeZeroedDueToX = 0;
	if(pNewRadMultiE->WfrEdgeCorrShouldBeDone)
	{
		if((xAbs < pNewRadMultiE->xWfrMin - DistAbsTol) || (xAbs > pNewRadMultiE->xWfrMax + DistAbsTol)) FieldShouldBeZeroedDueToX = 1;
	}

	int ixcOld = int((xAbs - pOldRadSingleE->xStart)*xStepInvOld + 1.E-06);
	if((ixcOld < 0) || (ixcOld > nx_mi_1Old))
	{
		FieldShouldBeZeroedDueToX = 1;
	}
	char FieldShouldBeZeroed = (FieldShouldBeZeroedDueToX || FieldShouldBeZeroedDueToZ);

	if(FieldShouldBeZeroed)
	{
		//*BufF = 0.; *(BufF+1) = 0.;
		if(TreatPolCompX)
		{
			*pEX_New = 0.;
			*(pEX_New+1) = 0.;
		}
		if(TreatPolCompZ)
		{
			*pEZ_New = 0.;
			*(pEZ_New+1) = 0.;
		}
		return;
	}

	double xRel = xAbs - (pOldRadSingleE->xStart + ixcOld*pOldRadSingleE->xStep);

	if(ixcOld == nx_mi_1Old) { ixStOld = ixcOld - 3; xRel += 2.*pOldRadSingleE->xStep;}
	else if(ixcOld == nx_mi_2Old) { ixStOld = ixcOld - 2; xRel += pOldRadSingleE->xStep;}
	else if(ixcOld == 0) { ixStOld = ixcOld; xRel -= pOldRadSingleE->xStep;}
	else ixStOld = ixcOld - 1;

	xRel *= xStepInvOld;
	int ixcOld_mi_ixStOld = ixcOld - ixStOld;

	if((izStOld != izStOldPrev) || (ixStOld != ixStOldPrev))
	{
		UseLowOrderInterp_PolCompX = 0, UseLowOrderInterp_PolCompZ = 0;
		//long TotOffsetOld = izStOld*PerZ_Old + ixStOld*PerX_Old + Two_ie;
		//long TotOffsetOld = izStOld*PerZ_Old + ixStOld*PerX_Old; //old is single slice
		long long TotOffsetOld = izStOld*PerZ_Old + ixStOld*PerX_Old; //old is single slice

		if(TreatPolCompX)
		{
			float* pExSt_Old = pOldRadSingleE->pBaseRadX + TotOffsetOld;
			srTGenOptElem::GetCellDataForInterpol(pExSt_Old, PerX_Old, PerZ_Old, AuxF);
			srTGenOptElem::SetupCellDataI(AuxF, AuxFI);
			UseLowOrderInterp_PolCompX = srTGenOptElem::CheckForLowOrderInterp(AuxF, AuxFI, ixcOld_mi_ixStOld, izcOld_mi_izStOld, &InterpolAux01, InterpolAux02, InterpolAux02I);

			if(!UseLowOrderInterp_PolCompX)
			{
				for(int i=0; i<2; i++) 
				{
					srTGenOptElem::SetupInterpolAux02(AuxF + i, &InterpolAux01, InterpolAux02 + i);
				}
				srTGenOptElem::SetupInterpolAux02(AuxFI, &InterpolAux01, InterpolAux02I);
			}
		}
		if(TreatPolCompZ)
		{
			float* pEzSt_Old = pOldRadSingleE->pBaseRadZ + TotOffsetOld;
			srTGenOptElem::GetCellDataForInterpol(pEzSt_Old, PerX_Old, PerZ_Old, AuxF+2);
			srTGenOptElem::SetupCellDataI(AuxF+2, AuxFI+1);
			UseLowOrderInterp_PolCompZ = srTGenOptElem::CheckForLowOrderInterp(AuxF+2, AuxFI+1, ixcOld_mi_ixStOld, izcOld_mi_izStOld, &InterpolAux01, InterpolAux02+2, InterpolAux02I+1);

			if(!UseLowOrderInterp_PolCompZ)
			{
				for(int i=0; i<2; i++) 
				{
					srTGenOptElem::SetupInterpolAux02(AuxF+2+i, &InterpolAux01, InterpolAux02+2+i);
				}
				srTGenOptElem::SetupInterpolAux02(AuxFI+1, &InterpolAux01, InterpolAux02I+1);
			}
		}
		ixStOldPrev = ixStOld; izStOldPrev = izStOld;
	}

	if(TreatPolCompX)
	{
		if(UseLowOrderInterp_PolCompX) 
		{
			srTGenOptElem::InterpolF_LowOrder(InterpolAux02, xRel, zRel, BufF, 0);
			srTGenOptElem::InterpolFI_LowOrder(InterpolAux02I, xRel, zRel, BufFI, 0);
		}
		else
		{
			srTGenOptElem::InterpolF(InterpolAux02, xRel, zRel, BufF, 0);
			srTGenOptElem::InterpolFI(InterpolAux02I, xRel, zRel, BufFI, 0);
		}

		(*BufFI) *= AuxFI->fNorm;
		srTGenOptElem::ImproveReAndIm(BufF, BufFI);

		//if(FieldShouldBeZeroed)
		//{
		//	*BufF = 0.; *(BufF+1) = 0.;
		//}

		*pEX_New = *BufF;
		*(pEX_New+1) = *(BufF+1);
	}
	if(TreatPolCompZ)
	{
		if(UseLowOrderInterp_PolCompZ) 
		{
			srTGenOptElem::InterpolF_LowOrder(InterpolAux02, xRel, zRel, BufF, 2);
			srTGenOptElem::InterpolFI_LowOrder(InterpolAux02I, xRel, zRel, BufFI, 1);
		}
		else
		{
			srTGenOptElem::InterpolF(InterpolAux02, xRel, zRel, BufF, 2);
			srTGenOptElem::InterpolFI(InterpolAux02I, xRel, zRel, BufFI, 1);
		}

		(*(BufFI+1)) *= (AuxFI+1)->fNorm;
		srTGenOptElem::ImproveReAndIm(BufF+2, BufFI+1);

		//if(FieldShouldBeZeroed)
		//{
		//	*(BufF+2) = 0.; *(BufF+3) = 0.;
		//}

		*pEZ_New = *(BufF+2);
		*(pEZ_New+1) = *(BufF+3);
	}
}

int srTGenOptElem::ReInterpolateWfrSliceSingleE_GPU(srTSRWRadStructAccessData& oldRadSingleE, srTSRWRadStructAccessData& newRadMultiE, int ie, TGPUUsageArg* pGPU)
{
	oldRadSingleE.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, oldRadSingleE.pBaseRadX, 2 * oldRadSingleE.ne * oldRadSingleE.nx * oldRadSingleE.nz * sizeof(float));
	oldRadSingleE.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, oldRadSingleE.pBaseRadZ, 2 * oldRadSingleE.ne * oldRadSingleE.nx * oldRadSingleE.nz * sizeof(float));
	newRadMultiE.pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, newRadMultiE.pBaseRadX, 2 * newRadMultiE.ne * newRadMultiE.nx * newRadMultiE.nz * sizeof(float));
	newRadMultiE.pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, newRadMultiE.pBaseRadZ, 2 * newRadMultiE.ne * newRadMultiE.nx * newRadMultiE.nz * sizeof(float));

	CAuxGPU::EnsureDeviceMemoryReady(pGPU, oldRadSingleE.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, oldRadSingleE.pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, newRadMultiE.pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, newRadMultiE.pBaseRadZ);

	const int bs = 32;
	dim3 blocks(newRadMultiE.nx / bs + ((newRadMultiE.nx & (bs - 1)) != 0), newRadMultiE.nz, 1);
	dim3 threads(bs, 1);

	srTSRWRadStructAccessData* pOldRadSingleE_dev = NULL; //HG27072024
	pOldRadSingleE_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &oldRadSingleE, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pOldRadSingleE_dev);

	srTSRWRadStructAccessData* pNewRadMultiE_dev = NULL; //HG27072024
	pNewRadMultiE_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, &newRadMultiE, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pNewRadMultiE_dev);

	ReInterpolateWfrSliceSingleE_Kernel <<<blocks, threads >>> (pOldRadSingleE_dev, pNewRadMultiE_dev, ie);

	CAuxGPU::ToHostAndFree(pGPU, pNewRadMultiE_dev, sizeof(srTSRWRadStructAccessData), true);
	CAuxGPU::ToHostAndFree(pGPU, pOldRadSingleE_dev, sizeof(srTSRWRadStructAccessData), true);

	oldRadSingleE.pBaseRadX = (float*)CAuxGPU::ToHostAndFree(pGPU, oldRadSingleE.pBaseRadX, 2 * oldRadSingleE.ne * oldRadSingleE.nx * oldRadSingleE.nz * sizeof(float), true);
	oldRadSingleE.pBaseRadZ = (float*)CAuxGPU::ToHostAndFree(pGPU, oldRadSingleE.pBaseRadZ, 2 * oldRadSingleE.ne * oldRadSingleE.nx * oldRadSingleE.nz * sizeof(float), true);
	
	CAuxGPU::MarkUpdated(pGPU, newRadMultiE.pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, newRadMultiE.pBaseRadZ, true, false);
	newRadMultiE.pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, newRadMultiE.pBaseRadX);
	newRadMultiE.pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, newRadMultiE.pBaseRadZ);
	
	return 0;
}

__global__ void SetupRadSliceConstE_Kernel(srTSRWRadStructAccessData* pRadAccessData, long ie, float* pInEx, float* pInEz)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
	int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range

	if(ix >= pRadAccessData->nx) return;
	if(iz >= pRadAccessData->nz) return;

	float *pEx0 = pRadAccessData->pBaseRadX + iz * pRadAccessData->nx * pRadAccessData->ne * 2 + ix * pRadAccessData->ne * 2 + ie * 2;
	float *pEz0 = pRadAccessData->pBaseRadZ + iz * pRadAccessData->nx * pRadAccessData->ne * 2 + ix * pRadAccessData->ne * 2 + ie * 2;
	float *tInEx = pInEx + iz * pRadAccessData->nx * 2 + ix * 2;
	float *tInEz = pInEz + iz * pRadAccessData->nx * 2 + ix * 2;
	
	*pEx0 = *tInEx;
	*(pEx0+1) = *(tInEx+1);
	*pEz0 = *tInEz;
	*(pEz0+1) = *(tInEz+1);
}

int srTGenOptElem::SetupRadSliceConstE_GPU(srTSRWRadStructAccessData* pRadAccessData, long ie, float* pInEx, float* pInEz, TGPUUsageArg* pGPU)
{
	pRadAccessData->pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, pRadAccessData->pBaseRadX, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pRadAccessData->pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, pRadAccessData->pBaseRadZ, 2 * pRadAccessData->ne * pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pInEx = (float*)CAuxGPU::ToDevice(pGPU, pInEx, 2*pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	pInEz = (float*)CAuxGPU::ToDevice(pGPU, pInEz, 2*pRadAccessData->nx * pRadAccessData->nz * sizeof(float));
	
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData->pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData->pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pInEx);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pInEz);
	
	srTSRWRadStructAccessData* pRadAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, pRadAccessData, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pRadAccessData_dev);

	const int bs = 256;
	dim3 blocks(pRadAccessData->nx / bs + ((pRadAccessData->nx & (bs - 1)) != 0), pRadAccessData->nz, 1);
	dim3 threads(bs, 1);

	SetupRadSliceConstE_Kernel <<<blocks, threads >>> (pRadAccessData_dev, ie, pInEx, pInEz);

	CAuxGPU::ToHostAndFree(pGPU, pRadAccessData_dev, sizeof(srTSRWRadStructAccessData), true);
	
	CAuxGPU::MarkUpdated(pGPU, pRadAccessData->pBaseRadX, true, false);
	CAuxGPU::MarkUpdated(pGPU, pRadAccessData->pBaseRadZ, true, false);
	pRadAccessData->pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, pRadAccessData->pBaseRadX);
	pRadAccessData->pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, pRadAccessData->pBaseRadZ);
	
	CAuxGPU::ToHostAndFree(pGPU, pInEx, 2*pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);
	CAuxGPU::ToHostAndFree(pGPU, pInEz, 2*pRadAccessData->nx * pRadAccessData->nz * sizeof(float), true);

	return 0;
}

template<bool IsCoordRepres, bool IsFreqRepres>
__global__ void ComputeRadMoments_Kernel(srTSRWRadStructAccessData* pSRWRadStructAccessData, int* IndLims, double* SumsZ, int ie)
{
	int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
	int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range

	if (ix >= pSRWRadStructAccessData->nx) return;
	if (iz >= pSRWRadStructAccessData->nz) return;

	const double TwoPi = 3.141592653590*2.;
	const double FourPi = TwoPi*2.;
	const double Inv_eV_In_m = 1.239842E-06;

	float *fpX0 = pSRWRadStructAccessData->pBaseRadX;
	float *fpZ0 = pSRWRadStructAccessData->pBaseRadZ;
	bool ExIsOK = fpX0 != 0; //13112011
	bool EzIsOK = fpZ0 != 0;

	long long PerX = pSRWRadStructAccessData->ne << 1;
	long long PerZ = PerX*pSRWRadStructAccessData->nx;

	int nx_mi_1 = pSRWRadStructAccessData->nx - 1;
	int nz_mi_1 = pSRWRadStructAccessData->nz - 1;

	double ePh = pSRWRadStructAccessData->eStart + pSRWRadStructAccessData->eStep*ie; //This assumes wavefront in Time domain; Photon Energy in eV !

	if(!IsFreqRepres)
	{
		ePh = pSRWRadStructAccessData->avgPhotEn; //?? OC041108
	}

	long long Two_ie = ie << 1;
	
	double Lamb_d_FourPi = Inv_eV_In_m/(FourPi*ePh);
	double Lamb_m = Lamb_d_FourPi*FourPi;

	double FourPi_d_Lamb = 1./Lamb_d_FourPi;
	
	double LocRobsX = pSRWRadStructAccessData->RobsX; //OC030409
	if(LocRobsX == 0.) LocRobsX = 100.*Lamb_m;
	double LocRobsZ = pSRWRadStructAccessData->RobsZ;
	if(LocRobsZ == 0.) LocRobsZ = 100.*Lamb_m;

	double FourPi_d_Lamb_d_Rx = FourPi_d_Lamb/LocRobsX;
	double FourPi_d_Lamb_d_Rz = FourPi_d_Lamb/LocRobsZ;

	double FourPi_d_Lamb_d_Rx_xStep = pSRWRadStructAccessData->xStep*FourPi_d_Lamb_d_Rx;
	double FourPi_d_Lamb_d_Rz_zStep = pSRWRadStructAccessData->zStep*FourPi_d_Lamb_d_Rz;
	double TwoPi_d_Lamb_d_Rx_xStep = 0.5*FourPi_d_Lamb_d_Rx_xStep;
	double TwoPi_d_Lamb_d_Rz_zStep = 0.5*FourPi_d_Lamb_d_Rz_zStep;

	double TwoPi_d_Lamb_d_Rx_xStepE2 = TwoPi_d_Lamb_d_Rx_xStep*TwoPi_d_Lamb_d_Rx_xStep;
	double TwoPi_d_Lamb_d_Rz_zStepE2 = TwoPi_d_Lamb_d_Rz_zStep*TwoPi_d_Lamb_d_Rz_zStep;

	double ff[22];
	bool vertCoordInsidePowLim = ((iz >= IndLims[2]) && (iz <= IndLims[3]));
	//long izPerZ = iz*PerZ;
	long long izPerZ = iz*PerZ;
	float *fpX_StartForX = fpX0 + izPerZ;
	float *fpZ_StartForX = fpZ0 + izPerZ;

	double z = pSRWRadStructAccessData->zStart + iz*pSRWRadStructAccessData->zStep;
	bool horCoordInsidePowLim = ((ix >= IndLims[0]) && (ix <= IndLims[1]));
	bool coordInsidePowLim = vertCoordInsidePowLim && horCoordInsidePowLim;

	//long ixPerX_p_Two_ie = ix*PerX + Two_ie;
	long long ixPerX_p_Two_ie = ix*PerX + Two_ie;
	float *fpX = fpX_StartForX + ixPerX_p_Two_ie;
	float *fpZ = fpZ_StartForX + ixPerX_p_Two_ie;

	double ExRe = 0., ExIm = 0., EzRe = 0., EzIm = 0.;
	if(ExIsOK)
	{
		ExRe = *fpX;
		ExIm = *(fpX+1);
	}
	if(EzIsOK)
	{
		EzRe = *fpZ;
		EzIm = *(fpZ+1);
	}

	double x = pSRWRadStructAccessData->xStart + ix*pSRWRadStructAccessData->xStep;
	ff[0] = ExRe*ExRe + ExIm*ExIm; // NormX
	ff[11] = EzRe*EzRe + EzIm*EzIm; // NormZ

	ff[1] = x*ff[0]; // <x>
	ff[3] = z*ff[0]; // <z>
	ff[12] = x*ff[11]; // <x>
	ff[14] = z*ff[11]; // <z>

	if(coordInsidePowLim) //OC13112010
	{
		ff[5] = x*ff[1]; // <xx>
		ff[8] = z*ff[3]; // <zz>
		ff[16] = x*ff[12]; // <xx>
		ff[19] = z*ff[14]; // <zz>
	}
	else
	{
		ff[5] = 0.; // <xx>
		ff[8] = 0.; // <zz>
		ff[16] = 0.; // <xx>
		ff[19] = 0.; // <zz>
	}

	if(IsCoordRepres && (ix > 0))
	{
		float *fpX_Prev = fpX - PerX;
		float *fpZ_Prev = fpZ - PerX;

		double ExReM = 0., ExImM = 0., EzReM = 0., EzImM = 0.;
		if(ExIsOK)
		{
			ExReM = *fpX_Prev; ExImM = *(fpX_Prev+1);
		}
		if(EzIsOK)
		{
			EzReM = *fpZ_Prev; EzImM = *(fpZ_Prev+1);
		}

		double ExReP_mi_ExReM = ExRe - ExReM;
		double ExImP_mi_ExImM = ExIm - ExImM;
		double EzReP_mi_EzReM = EzRe - EzReM;
		double EzImP_mi_EzImM = EzIm - EzImM;

		double ExImP_mi_ExImM_ExRe_mi_ExReP_mi_ExReM_ExIm = ExImP_mi_ExImM*ExRe - ExReP_mi_ExReM*ExIm;
		ff[2] = ExImP_mi_ExImM_ExRe_mi_ExReP_mi_ExReM_ExIm + TwoPi_d_Lamb_d_Rx_xStep*x*ff[0]; // <x'>

		double EzImP_mi_EzImM_EzRe_mi_EzReP_mi_EzReM_EzIm = EzImP_mi_EzImM*EzRe - EzReP_mi_EzReM*EzIm;
		ff[13] = EzImP_mi_EzImM_EzRe_mi_EzReP_mi_EzReM_EzIm + TwoPi_d_Lamb_d_Rx_xStep*x*ff[11]; // <x'>

		if(coordInsidePowLim) //OC13112010
		{
			ff[6] = x*ff[2]; // <xx'>
			ff[7] = (ExReP_mi_ExReM*ExReP_mi_ExReM + ExImP_mi_ExImM*ExImP_mi_ExImM) 
					+ ExImP_mi_ExImM_ExRe_mi_ExReP_mi_ExReM_ExIm*TwoPi_d_Lamb_d_Rx_xStep*x
					+ TwoPi_d_Lamb_d_Rx_xStepE2*x*x*ff[0]; // <x'x'>
			ff[17] = x*ff[13]; // <xx'>
			ff[18] = EzReP_mi_EzReM*EzReP_mi_EzReM + EzImP_mi_EzImM*EzImP_mi_EzImM
					+ EzImP_mi_EzImM_EzRe_mi_EzReP_mi_EzReM_EzIm*TwoPi_d_Lamb_d_Rx_xStep*x
					+ TwoPi_d_Lamb_d_Rx_xStepE2*x*x*ff[11]; // <x'x'>
		}
		else
		{
			ff[6] = 0.; // <xx'>
			ff[7] = 0.; // <x'x'>
			ff[17] = 0.; // <xx'>
			ff[18] = 0.; // <x'x'>
		}
	}
	else
	{
		ff[2] = 0.; // <x'>
		ff[6] = 0.; // <xx'>
		ff[7] = 0.; // <x'x'>
		ff[13] = 0.; // <x'>
		ff[17] = 0.; // <xx'>
		ff[18] = 0.; // <x'x'>
	}

	if(IsCoordRepres && (iz > 0))
	{
		float *fpX_Prev = fpX - PerZ;
		float *fpZ_Prev = fpZ - PerZ;

		double ExReM = 0., ExImM = 0, EzReM = 0., EzImM = 0.;
		if(ExIsOK)
		{
			ExReM = *fpX_Prev; ExImM = *(fpX_Prev+1);
		}
		if(EzIsOK)
		{
			EzReM = *fpZ_Prev; EzImM = *(fpZ_Prev+1);
		}

		double ExReP_mi_ExReM = ExRe - ExReM;
		double ExImP_mi_ExImM = ExIm - ExImM;
		double EzReP_mi_EzReM = EzRe - EzReM;
		double EzImP_mi_EzImM = EzIm - EzImM;

		double ExImP_mi_ExImM_ExRe_mi_ExReP_mi_ExReM_ExIm = ExImP_mi_ExImM*ExRe - ExReP_mi_ExReM*ExIm;
		ff[4] = ExImP_mi_ExImM_ExRe_mi_ExReP_mi_ExReM_ExIm + TwoPi_d_Lamb_d_Rz_zStep*z*ff[0]; // <z'>

		double EzImP_mi_EzImM_EzRe_mi_EzReP_mi_EzReM_EzIm = EzImP_mi_EzImM*EzRe - EzReP_mi_EzReM*EzIm;
		ff[15] = EzImP_mi_EzImM_EzRe_mi_EzReP_mi_EzReM_EzIm + TwoPi_d_Lamb_d_Rz_zStep*z*ff[11]; // <z'>

		if(coordInsidePowLim) //OC13112010
		{
			ff[9] = z*ff[4]; // <zz'>
			ff[10] = ExReP_mi_ExReM*ExReP_mi_ExReM + ExImP_mi_ExImM*ExImP_mi_ExImM
					+ ExImP_mi_ExImM_ExRe_mi_ExReP_mi_ExReM_ExIm*TwoPi_d_Lamb_d_Rz_zStep*z
					+ TwoPi_d_Lamb_d_Rz_zStepE2*z*z*ff[0]; // <z'z'>
			ff[20] = z*ff[15]; // <zz'>
			ff[21] = EzReP_mi_EzReM*EzReP_mi_EzReM + EzImP_mi_EzImM*EzImP_mi_EzImM
					+ EzImP_mi_EzImM_EzRe_mi_EzReP_mi_EzReM_EzIm*TwoPi_d_Lamb_d_Rz_zStep*z
					+ TwoPi_d_Lamb_d_Rz_zStepE2*z*z*ff[11]; // <z'z'>
		}
		else
		{
			ff[9] = 0.; // <zz'>
			ff[10] = 0.; // <z'z'>
			ff[20] = 0.; // <zz'>
			ff[21] = 0.;
		}
	}
	else
	{
		ff[4] = 0.; // <z'>
		ff[9] = 0.; // <zz'>
		ff[10] = 0.; // <z'z'>
		ff[15] = 0.; // <z'>
		ff[20] = 0.; // <zz'>
		ff[21] = 0.; // <z'z'>
	}

	if((ix == 0) || (ix == nx_mi_1)) for(int k=0; k<22; k++) ff[k] *= 0.5;
	if(ix == 1)
	{
		ff[2] *= 0.5; // <x'>
		ff[6] *= 0.5; // <xx'>
		ff[7] *= 0.5; // <x'x'>
		ff[13] *= 0.5; // <x'>
		ff[17] *= 0.5; // <xx'>
		ff[18] *= 0.5; // <x'x'>
	}

	cg::coalesced_group g = cg::coalesced_threads();
	for(int kk=0; kk<22; kk++)
		ff[kk] = cg::reduce(g, ff[kk], cg::plus<double>());

	if(g.thread_rank() == 0)
	{
		if((iz == 0) || (iz == nz_mi_1)) for(int k2=0; k2<22; k2++) ff[k2] *= 0.5;
		if(iz == 1)
		{
			ff[4] *= 0.5; // <z'>
			ff[9] *= 0.5; // <zz'>
			ff[10] *= 0.5; // <z'z'>
			ff[15] *= 0.5; // <z'>
			ff[20] *= 0.5; // <zz'>
			ff[21] *= 0.5; // <z'z'>
		}

		//printf("%f\n", ff[0]);
		for (int kk = 0; kk < 22; kk++) atomicAdd(SumsZ + kk, ff[kk]);
	}
}

void srTGenOptElem::ComputeRadMoments_GPU(srTSRWRadStructAccessData* pSRWRadStructAccessData, int ie, double* SumsZ, int* IndLims, TGPUUsageArg* pGPU) //HG26072024
{
	pSRWRadStructAccessData->pBaseRadX = (float*)CAuxGPU::ToDevice(pGPU, pSRWRadStructAccessData->pBaseRadX, 2 * pSRWRadStructAccessData->ne * pSRWRadStructAccessData->nx * pSRWRadStructAccessData->nz * sizeof(float));
	pSRWRadStructAccessData->pBaseRadZ = (float*)CAuxGPU::ToDevice(pGPU, pSRWRadStructAccessData->pBaseRadZ, 2 * pSRWRadStructAccessData->ne * pSRWRadStructAccessData->nx * pSRWRadStructAccessData->nz * sizeof(float));
	SumsZ = (double*)CAuxGPU::ToDevice(pGPU, SumsZ, 22 * sizeof(double), true, false, 2);
	IndLims = (int*)CAuxGPU::ToDevice(pGPU, IndLims, 4 * sizeof(int));
	
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pSRWRadStructAccessData->pBaseRadX);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pSRWRadStructAccessData->pBaseRadZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, SumsZ);
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, IndLims);
	
	srTSRWRadStructAccessData* pSRWRadStructAccessData_dev = (srTSRWRadStructAccessData*)CAuxGPU::ToDevice(pGPU, pSRWRadStructAccessData, sizeof(srTSRWRadStructAccessData));
	CAuxGPU::EnsureDeviceMemoryReady(pGPU, pSRWRadStructAccessData_dev);

	const int bs = 256;
	dim3 blocks((pSRWRadStructAccessData->nx - 1) / bs + 1, pSRWRadStructAccessData->nz, 1);
	dim3 threads(bs, 1);

	bool IsCoordRepres = (pSRWRadStructAccessData->Pres == 0);
	bool IsFreqRepres = (pSRWRadStructAccessData->PresT == 0);

	if (IsCoordRepres && IsFreqRepres) ComputeRadMoments_Kernel<true, true> <<<blocks, threads >>> 			(pSRWRadStructAccessData_dev, IndLims, SumsZ, ie);
	else if (IsCoordRepres && !IsFreqRepres) ComputeRadMoments_Kernel<true, false> <<<blocks, threads >>>   (pSRWRadStructAccessData_dev, IndLims, SumsZ, ie);
	else if (!IsCoordRepres && IsFreqRepres) ComputeRadMoments_Kernel<false, true> <<<blocks, threads >>>   (pSRWRadStructAccessData_dev, IndLims, SumsZ, ie);
	else if (!IsCoordRepres && !IsFreqRepres) ComputeRadMoments_Kernel<false, false> <<<blocks, threads >>> (pSRWRadStructAccessData_dev, IndLims, SumsZ, ie);

	CAuxGPU::ToHostAndFree(pGPU, pSRWRadStructAccessData_dev, sizeof(srTSRWRadStructAccessData), true);
	
	pSRWRadStructAccessData->pBaseRadX = (float*)CAuxGPU::GetHostPtr(pGPU, pSRWRadStructAccessData->pBaseRadX);
	pSRWRadStructAccessData->pBaseRadZ = (float*)CAuxGPU::GetHostPtr(pGPU, pSRWRadStructAccessData->pBaseRadZ);
	
	CAuxGPU::MarkUpdated(pGPU, SumsZ, true, false);

	CAuxGPU::ToHostAndFree(pGPU, IndLims, 4 * sizeof(int), true);
	CAuxGPU::ToHostAndFree(pGPU, SumsZ, 22 * sizeof(double));
}
#endif