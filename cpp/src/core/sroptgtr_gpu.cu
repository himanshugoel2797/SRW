/************************************************************************//**
 * File: sroptgtr_gpu.cu
 * Description: Optical element: Transmission (CUDA implementation)
 * Project: Synchrotron Radiation Workshop
 * First release: 2023
 *
 * Copyright (C) Brookhaven National Laboratory
 * All Rights Reserved
 *
 * @author H.Goel
 * @version 1.0
 ***************************************************************************/

#include "sroptgtr.h"
#ifdef _OFFLOAD_GPU
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_math_constants.h"

#include <stdio.h>
#include <iostream>
#include <chrono>

int srTGenTransmission::RadPointModifierParallel(srTSRWRadStructAccessData* pRadAccessData, void* pBufVars, long pBufVarsSz, gpuUsageArg *pGpuUsage)
{
    GenTransNumData.pData = (char*)AuxGpu::ToDevice(pGpuUsage, GenTransNumData.pData, GenTransNumData.DimSizes[0] * (int)GenTransNumData.DimSizes[1] * (int)GenTransNumData.DimSizes[2] * sizeof(double) * 2);
	AuxGpu::EnsureDeviceMemoryReady(pGpuUsage, GenTransNumData.pData);
    int retCode = RadPointModifierParallelImpl<srTGenTransmission>(pRadAccessData, pBufVars, pBufVarsSz, this, pGpuUsage); 
	GenTransNumData.pData = (char*)AuxGpu::ToHostAndFree(pGpuUsage, GenTransNumData.pData, GenTransNumData.DimSizes[0] * (int)GenTransNumData.DimSizes[1] * (int)GenTransNumData.DimSizes[2] * sizeof(double) * 2, true);
    return retCode;
} //HG03092022
#endif