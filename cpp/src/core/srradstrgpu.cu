
#include <hip/hip_runtime.h>
#ifdef _OFFLOAD_GPU
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "math_constants.h"
#include <stdio.h>
#include <iostream>
#include <chrono>


__global__ void MultiplyElFieldByPhaseLin_Kernel(double xMult, double zMult, float* pBaseRadX, float* pBaseRadZ, int nWfr, int nz, int nx, int ne, float zStart, float zStep, float xStart, float xStep) {
    int ix = (blockIdx.x * blockDim.x + threadIdx.x); //nx range
    int iz = (blockIdx.y * blockDim.y + threadIdx.y); //nz range
    int iwfr = (blockIdx.z * blockDim.z + threadIdx.z); //nWfr range

    if (ix < nx && iz < nz && iwfr < nWfr) 
    {
		bool RadXisDefined = (pBaseRadX != 0);
		bool RadZisDefined = (pBaseRadZ != 0);

		double z = zStart + iz * zStep;
		double x = xStart + ix * xStep;
		double dPhZ = zMult * z;
		double dPh = dPhZ + xMult * x;
		double cosPh, sinPh;
		sincos(dPh, &sinPh, &cosPh);

		long long offset = iwfr * nz * nx * ne * 2 + iz * nx * ne * 2 + ix * ne * 2;
		float* tEx = pBaseRadX + offset;
		float* tEz = pBaseRadZ + offset;
		for (int ie = 0; ie < ne; ie++)
		{
			if (RadXisDefined)
			{
				//*(tEx++) *= a; *(tEx++) *= a;
				double newReEx = (*tEx) * cosPh - (*(tEx + 1)) * sinPh;
				double newImEx = (*tEx) * sinPh + (*(tEx + 1)) * cosPh;
				*(tEx++) = (float)newReEx; *(tEx++) = (float)newImEx;
			}
			if (RadZisDefined)
			{
				//*(tEz++) *= a; *(tEz++) *= a;
				double newReEz = (*tEz) * cosPh - (*(tEz + 1)) * sinPh;
				double newImEz = (*tEz) * sinPh + (*(tEz + 1)) * cosPh;
				*(tEz++) = (float)newReEz; *(tEz++) = (float)newImEz;
			}
		}
    }
}

void MultiplyElFieldByPhaseLin_GPU(double xMult, double zMult, float* pBaseRadX, float* pBaseRadZ, int nWfr, int nz, int nx, int ne, float zStart, float zStep, float xStart, float xStep)
{
    const int bs = 256;
    dim3 blocks(nx / bs + ((nx & (bs - 1)) != 0), nz, nWfr);
    dim3 threads(bs, 1);
    MultiplyElFieldByPhaseLin_Kernel<< <blocks, threads >> > (xMult, zMult, pBaseRadX, pBaseRadZ, nWfr, nz, nx, ne, zStart, zStep, xStart, xStep);

#ifdef _DEBUG
	cudaStreamSynchronize(0);
    auto err = cudaGetLastError();
    printf("%s\r\n", cudaGetErrorString(err));
#endif
}

#endif